#include "hip/hip_runtime.h"
#include "cuMatrix.h"

__global__ void make_BlackWhite(int *image, int N){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	image[y*N + x] = image[y*N + x] > 128 ? 255 : 0;
}

void convertToArray(int **matrix, int *array, int N){
	for(unsigned int i=0; i< N; i++)
		for(unsigned int j=0; j< N; j++)
			array[i*N+ j] = matrix[i][j];
}
